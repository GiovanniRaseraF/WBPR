#include"../include/parallel_graph.cuh"

void push_relabel(ull *V, ull *gpu_V, ull source, ull sink, int *cpu_height, int *cpu_excess_flow, int *cpu_adjmtx, int *cpu_rflowmtx, int *Excess_total, int *gpu_height, int *gpu_excess_flow, int *gpu_adjmtx, int *gpu_rflowmtx)
{
    /* Instead of checking for overflowing vertices(as in the sequential push relabel),
     * sum of excess flow values of sink and source are compared against Excess_total 
     * If the sum is lesser than Excess_total, 
     * it means that there is atleast one more vertex with excess flow > 0, apart from source and sink
     */

    /* declaring the mark and scan boolean arrays used in the global_relabel routine outside the while loop 
     * This is not to lose the mark values if it goes out of scope and gets redeclared in the next iteration 
     */

    bool *mark,*scanned;
    mark = (bool*)malloc(*V*sizeof(bool));
    scanned = (bool*)malloc(*V*sizeof(bool));

    // initialising mark values to false for all nodes
    for(ull i = 0; i < *V; i++)
    {
        mark[i] = false;
    }

    while((cpu_excess_flow[source] + cpu_excess_flow[sink]) < *Excess_total)
    {
        // printf("cpu_excess_flow[source]: %d, cpu_excess_flow[sink]: %d\n",cpu_excess_flow[source], cpu_excess_flow[sink]);
        // copying height values to CUDA device global memory
        CHECK(hipMemcpy(gpu_height,cpu_height,*V*sizeof(int),hipMemcpyHostToDevice));

        printf("Invoking kernel\n");

        // invoking the push_relabel_kernel
        push_relabel_kernel<<<1,2>>>(gpu_V,source,sink,gpu_height,gpu_excess_flow,gpu_adjmtx,gpu_rflowmtx);

        hipDeviceSynchronize();

        hipError_t error = hipGetLastError();
        if (error != hipSuccess) {
            // Handle error
            fprintf(stderr, "Kernel launching error: %s\n", hipGetErrorString(error));
        }

        // copying height, excess flow and residual flow values from device to host memory
        CHECK(hipMemcpy(cpu_height,gpu_height,*V*sizeof(int),hipMemcpyDeviceToHost));
        CHECK(hipMemcpy(cpu_excess_flow,gpu_excess_flow,*V*sizeof(int),hipMemcpyDeviceToHost));
        CHECK(hipMemcpy(cpu_rflowmtx,gpu_rflowmtx,*V**V*sizeof(int),hipMemcpyDeviceToHost));
        
        printf("After invoking\n");
        //print(V,cpu_height,cpu_excess_flow,cpu_rflowmtx,cpu_adjmtx);
        printf("Excess total : %d\n",*Excess_total);
        // perform the global_relabel routine on host
        global_relabel(V,source,sink,cpu_height,cpu_excess_flow,cpu_adjmtx,cpu_rflowmtx,Excess_total,mark,scanned);

        printf("\nAfter global relabel\n");
        //print(V,cpu_height,cpu_excess_flow,cpu_rflowmtx,cpu_adjmtx);
        printf("Excess total : %d\n",*Excess_total);
    }

}
