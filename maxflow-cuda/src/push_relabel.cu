#include"../include/parallel_graph.cuh"

bool compare_excess_flow(int *new_excess_flow, int *old_excess_flow, int V)
{
    for(int i = 0; i < V; i++)
    {
        if (new_excess_flow[i] != old_excess_flow[i])
        {
            return false;
        }
    }
    return true;
}

void copy_excess_flow(int *new_excess_flow, int *old_excess_flow, int V)
{
    for(int i = 0; i < V; i++)
    {
        old_excess_flow[i] = new_excess_flow[i];
    }
}

void printExcessFlow(int V, int *excess_flow)
{
    printf("Excess flow values : \n");
    for(int i = 0; i < V; i++)
    {
        printf("%d ",excess_flow[i]);
    }
    printf("\n");
}


void push_relabel(int V, int source, int sink, int *cpu_height, int *cpu_excess_flow, int *cpu_adjmtx, int *cpu_rflowmtx, int *Excess_total, int *gpu_height, int *gpu_excess_flow, int *gpu_adjmtx, int *gpu_rflowmtx)
{
    /* Instead of checking for overflowing vertices(as in the sequential push relabel),
     * sum of excess flow values of sink and source are compared against Excess_total 
     * If the sum is lesser than Excess_total, 
     * it means that there is atleast one more vertex with excess flow > 0, apart from source and sink
     */

    /* declaring the mark and scan boolean arrays used in the global_relabel routine outside the while loop 
     * This is not to lose the mark values if it goes out of scope and gets redeclared in the next iteration 
     */
    
    bool *mark,*scanned;
    mark = (bool*)malloc(V*sizeof(bool));
    scanned = (bool*)malloc(V*sizeof(bool));
    bool first_iteration = true;
    int *cpu_excess_flow_old;
    cpu_excess_flow_old = (int*)malloc(V*sizeof(int));

    // initialising mark values to false for all nodes
    for(int i = 0; i < V; i++)
    {
        mark[i] = false;
    }

    while((cpu_excess_flow[source] + cpu_excess_flow[sink]) < *Excess_total)
    {
        if (!first_iteration)
        {
            copy_excess_flow(cpu_excess_flow,cpu_excess_flow_old,V);
        }

        printf("cpu_excess_flow[source]: %d, cpu_excess_flow[sink]: %d\n",cpu_excess_flow[source], cpu_excess_flow[sink]);
        // copying height values to CUDA device global memory
        hipMemcpy(gpu_height,cpu_height,V*sizeof(int),hipMemcpyHostToDevice);

        //printf("Invoking kernel\n");

        // invoking the push_relabel_kernel
        push_relabel_kernel<<<number_of_blocks_nodes,threads_per_block>>>(V,source,sink,gpu_height,gpu_excess_flow,gpu_adjmtx,gpu_rflowmtx);

        hipDeviceSynchronize();


        // copying height, excess flow and residual flow values from device to host memory
        hipMemcpy(cpu_height,gpu_height,V*sizeof(int),hipMemcpyDeviceToHost);
        hipMemcpy(cpu_excess_flow,gpu_excess_flow,V*sizeof(int),hipMemcpyDeviceToHost);
        hipMemcpy(cpu_rflowmtx,gpu_rflowmtx,V*V*sizeof(int),hipMemcpyDeviceToHost);

        //printf("After invoking\n");
        //print(V,cpu_height,cpu_excess_flow,cpu_rflowmtx,cpu_adjmtx);
        //printf("Excess total : %d\n",*Excess_total);
        // perform the global_relabel routine on host
        // printf("Before global relabel, Excess total : %d\n",*Excess_total);
        global_relabel(V,source,sink,cpu_height,cpu_excess_flow,cpu_adjmtx,cpu_rflowmtx,Excess_total,mark,scanned);

        // printf("After global relabel, Excess total : %d\n",*Excess_total);

        //printf("\nAfter global relabel\n");
        //print(V,cpu_height,cpu_excess_flow,cpu_rflowmtx,cpu_adjmtx);
        //printf("Excess total : %d\n",*Excess_total);
        first_iteration = false;

        // if (compare_excess_flow(cpu_excess_flow,cpu_excess_flow_old,V))
        // {
        //     printf("Same excess flow!!!!!!\n");
        //     printExcessFlow(V,cpu_excess_flow);
        //     exit(1);
        // }
        printf("Excess total : %d\n",*Excess_total);
        printExcessFlow(V,cpu_excess_flow);
    }

}
