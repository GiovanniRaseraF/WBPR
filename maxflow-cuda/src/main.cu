#include"../include/parallel_graph.cuh"
#include"../include/serial_graph.h"
#include "../include/graph.h"
#include "../include/utils.cuh"

int main(int argc, char **argv)
{
    // checking if sufficient number of arguments (4) are passed in CLI
    if(argc != 4)
    {
        printf("Invalid number of arguments passed during execution\n");
        exit(0);
    }
    // reading the arguments passed in CLI
    char* filename = argv[1];
    // int V = atoi(argv[2]);
    // int E = atoi(argv[3]);
    int source = atoi(argv[2]);
    int sink = atoi(argv[3]);

    // Read from snap txt
    CSRGraph csr_graph;
    csr_graph.buildFromDIMACSFile(filename);

    ResidualGraph res_graph;
    res_graph.buildFromCSRGraph(csr_graph);

    printf("Reading graph from file %s\n",filename);
    
    int V = csr_graph.num_nodes;
    int E = csr_graph.num_edges;
    source = csr_graph.source_node;
    sink = csr_graph.sink_node;

    printf("Source: %d, Sink: %d\n", source, sink);

    // Print res_graph
    // res_graph.print();


    // declaring variables to store graph data on host as well as on CUDA device global memory 
    int *cpu_height,*gpu_height;
    int *cpu_excess_flow,*gpu_excess_flow;
    int *Excess_total;
    int *gpu_destinations, *gpu_rdestinations;
    int *gpu_offsets, *gpu_roffsets;
    int *gpu_capcities;
    int *gpu_fflows, *gpu_bflows; // Forward and backward flows
    int *gpu_flow_idx; // Index of the flow
    int *cpu_avq, *gpu_avq;
    int cycle = res_graph.num_nodes;
    int *gpu_cycle;


    
    // allocating host memory
    cpu_height = (int*)malloc(V*sizeof(int));
    cpu_excess_flow = (int*)malloc(V*sizeof(int));
    Excess_total = (int*)malloc(sizeof(int));
    cpu_avq = (int*)malloc(V*sizeof(int));

    for (int i = 0; i < V; i++)
    {
        cpu_avq[i] = 0;
    }





    // allocating CUDA device global memory
    CHECK(hipMalloc((void**)&gpu_height, V*sizeof(int)));
    CHECK(hipMalloc((void**)&gpu_excess_flow, V*sizeof(int)));
    CHECK(hipMalloc((void**)&gpu_destinations,E*sizeof(int)));
    CHECK(hipMalloc((void**)&gpu_offsets, (V+1)*sizeof(int)));
    CHECK(hipMalloc((void**)&gpu_capcities, E*sizeof(int)));
    CHECK(hipMalloc((void**)&gpu_fflows, E*sizeof(int)));
    CHECK(hipMalloc((void**)&gpu_rdestinations,E*sizeof(int)));
    CHECK(hipMalloc((void**)&gpu_roffsets, (V+1)*sizeof(int)));
    CHECK(hipMalloc((void**)&gpu_bflows, E*sizeof(int)));
    CHECK(hipMalloc((void**)&gpu_flow_idx, E*sizeof(int)));
    CHECK(hipMalloc((void**)&gpu_avq, V*sizeof(int)));
    CHECK(hipMalloc((void**)&gpu_cycle, sizeof(int)));


    // readgraph
    // readgraph(filename,V,E,source,sink,cpu_height,cpu_excess_flow,cpu_adjmtx,cpu_rflowmtx);
    

    // time start
    printf("Starting preflow\n");

    // invoking the preflow function to initialise values in host
    preflow(V,source,sink,cpu_height,cpu_excess_flow, 
            (res_graph.offsets), (res_graph.destinations), (res_graph.capacities), (res_graph.forward_flows), (res_graph.backward_flows),
            (res_graph.roffsets), (res_graph.rdestinations), (res_graph.flow_index), Excess_total);
    
    printf("Excess_total: %d\n",*Excess_total);


    //print(V,cpu_height,cpu_excess_flow,cpu_rflowmtx,cpu_adjmtx);

    // copying host data to CUDA device global memory
    CHECK(hipMemcpy(gpu_height,cpu_height,V*sizeof(int),hipMemcpyHostToDevice));
    CHECK(hipMemcpy(gpu_excess_flow,cpu_excess_flow,V*sizeof(int),hipMemcpyHostToDevice));
    CHECK(hipMemcpy(gpu_offsets, res_graph.offsets, (res_graph.num_nodes + 1)*sizeof(int), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(gpu_destinations, res_graph.destinations, res_graph.num_edges*sizeof(int), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(gpu_capcities, res_graph.capacities, res_graph.num_edges*sizeof(int), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(gpu_fflows, res_graph.forward_flows, res_graph.num_edges*sizeof(int), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(gpu_roffsets, res_graph.roffsets, (res_graph.num_nodes + 1)*sizeof(int), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(gpu_rdestinations, res_graph.rdestinations, res_graph.num_edges*sizeof(int), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(gpu_bflows, res_graph.backward_flows, res_graph.num_edges*sizeof(int), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(gpu_flow_idx, res_graph.flow_index, res_graph.num_edges*sizeof(int), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(gpu_avq, cpu_avq, res_graph.num_nodes*sizeof(int), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(gpu_cycle, &cycle, sizeof(int), hipMemcpyHostToDevice));
    //hipMemcpy(gpu_adjmtx,cpu_adjmtx,V*V*sizeof(int),hipMemcpyHostToDevice);
    // hipMemcpy(gpu_rflowmtx,cpu_rflowmtx,V*V*sizeof(int),hipMemcpyHostToDevice);

    printf("Starting push_relabel\n");

    // push_relabel()
    push_relabel(V,E,source,sink,cpu_height,cpu_excess_flow, 
                res_graph.offsets, res_graph.destinations, res_graph.capacities, res_graph.forward_flows, res_graph.backward_flows, 
                res_graph.roffsets, res_graph.rdestinations, res_graph.flow_index,
                Excess_total,
                gpu_height, gpu_excess_flow,
                gpu_offsets, gpu_destinations, gpu_capcities, gpu_fflows, gpu_bflows,
                gpu_roffsets, gpu_rdestinations, gpu_flow_idx, gpu_avq, gpu_cycle);
    
    // store value from serial implementation
    //int serial_check = check(V,E,source,sink);

    // print values from both implementations
    printf("The maximum flow value of this flow network as calculated by the parallel implementation is %d, %d\n",cpu_excess_flow[sink], *Excess_total);
    //printf("The maximum flow of this flow network as calculated by the serial implementation is %d\n",serial_check);
    
    // print correctness check result
    // if(cpu_excess_flow[sink] == serial_check)
    // {
    //     printf("Passed correctness check\n");
    // }
    // else
    // {
    //     printf("Failed correctness check\n");
    // }

    // free device memory
    CHECK(hipFree(gpu_height));
    CHECK(hipFree(gpu_excess_flow));
    CHECK(hipFree(gpu_offsets));
    CHECK(hipFree(gpu_destinations));
    CHECK(hipFree(gpu_capcities));
    CHECK(hipFree(gpu_fflows));
    CHECK(hipFree(gpu_bflows));
    CHECK(hipFree(gpu_roffsets));
    CHECK(hipFree(gpu_rdestinations));
    CHECK(hipFree(gpu_flow_idx));
    CHECK(hipFree(gpu_avq));

    //hipFree(gpu_adjmtx);
    //hipFree(gpu_rflowmtx);
    
    // free host memory
    free(cpu_height);
    free(cpu_excess_flow);
    free(Excess_total);
    free(cpu_avq);
    //free(cpu_adjmtx);
    //free(cpu_rflowmtx);
    
    // return 0 and end program
    return 0;

}
