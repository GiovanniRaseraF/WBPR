#include"../include/parallel_graph.cuh"
#include"../include/serial_graph.h"
#include "../include/graph.h"

int main(int argc, char **argv)
{
    // checking if sufficient number of arguments (4) are passed in CLI
    if(argc != 4)
    {
        printf("Invalid number of arguments passed during execution\n");
        exit(0);
    }
    // reading the arguments passed in CLI
    char* filename = argv[1];
    // int V = atoi(argv[2]);
    // int E = atoi(argv[3]);
    int source = atoi(argv[2]);
    int sink = atoi(argv[3]);

    // Read from snap txt
    CSRGraph csr_graph;
    csr_graph.buildFromTxtFile(filename);

    ResidualGraph res_graph;
    res_graph.buildFromCSRGraph(csr_graph);

    printf("Reading graph from file %s\n",filename);
    
    int V = csr_graph.num_nodes;
    int E = csr_graph.num_edges;

    // Print res_graph
    res_graph.print();


    // declaring variables to store graph data on host as well as on CUDA device global memory 
    int *cpu_height,*gpu_height;
    int *cpu_excess_flow,*gpu_excess_flow;
    int *Excess_total;
    int *gpu_destinations, *gpu_rdestinations;
    int *gpu_offsets, *gpu_roffsets;
    int *gpu_capcities, *gpu_rcapacities;

    
    // allocating host memory
    cpu_height = (int*)malloc(V*sizeof(int));
    cpu_excess_flow = (int*)malloc(V*sizeof(int));
    Excess_total = (int*)malloc(sizeof(int));




    // allocating CUDA device global memory
    hipMalloc((void**)&gpu_height,V*sizeof(int));
    hipMalloc((void**)&gpu_excess_flow,V*sizeof(int));
    hipMalloc((void**)&gpu_destinations,V*sizeof(int));
    hipMalloc((void**)&gpu_offsets,E*sizeof(int));
    hipMalloc((void**)&gpu_capcities,E*sizeof(int));
    hipMalloc((void**)&gpu_rdestinations,V*sizeof(int));
    hipMalloc((void**)&gpu_roffsets,E*sizeof(int));
    hipMalloc((void**)&gpu_rcapacities,E*sizeof(int));


    // readgraph
    // readgraph(filename,V,E,source,sink,cpu_height,cpu_excess_flow,cpu_adjmtx,cpu_rflowmtx);
    

    // time start
    printf("Starting preflow\n");

    // invoking the preflow function to initialise values in host
    preflow(V,source,sink,cpu_height,cpu_excess_flow, 
            &(res_graph.offsets), &(res_graph.destinations), &(res_graph.capacities), &(res_graph.flows),
            &(res_graph.roffsets), &(res_graph.rdestinations), &(res_graph.rflows), Excess_total);
    
    printf("Excess_total: %d\n",*Excess_total);

    // Print the result of preflow,
    printf("Preflow result:\n");
    printf("Flow: ");
    for (int i=0; i < res_graph.flows.size(); i++) {
        printf("%d ", res_graph.flows[i]);
    }
    printf("\n");
    printf("Rflow: ");
    for (int i=0; i < res_graph.rflows.size(); i++) {
        printf("%d ", res_graph.rflows[i]);
    }
    printf("\n");
    


    //print(V,cpu_height,cpu_excess_flow,cpu_rflowmtx,cpu_adjmtx);

    // copying host data to CUDA device global memory
    hipMemcpy(gpu_height,cpu_height,V*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(gpu_excess_flow,cpu_excess_flow,V*sizeof(int),hipMemcpyHostToDevice);
    //hipMemcpy(gpu_adjmtx,cpu_adjmtx,V*V*sizeof(int),hipMemcpyHostToDevice);
    // hipMemcpy(gpu_rflowmtx,cpu_rflowmtx,V*V*sizeof(int),hipMemcpyHostToDevice);

    // push_relabel()
    // push_relabel(V,source,sink,cpu_height,cpu_excess_flow,cpu_adjmtx,cpu_rflowmtx,Excess_total,gpu_height,gpu_excess_flow,gpu_adjmtx,gpu_rflowmtx);
    
    // store value from serial implementation
    int serial_check = check(V,E,source,sink);

    // print values from both implementations
    printf("The maximum flow value of this flow network as calculated by the parallel implementation is %d\n",cpu_excess_flow[sink]);
    printf("The maximum flow of this flow network as calculated by the serial implementation is %d\n",serial_check);
    
    // print correctness check result
    if(cpu_excess_flow[sink] == serial_check)
    {
        printf("Passed correctness check\n");
    }
    else
    {
        printf("Failed correctness check\n");
    }

    // free device memory
    hipFree(gpu_height);
    hipFree(gpu_excess_flow);
    //hipFree(gpu_adjmtx);
    //hipFree(gpu_rflowmtx);
    
    // free host memory
    free(cpu_height);
    free(cpu_excess_flow);
    //free(cpu_adjmtx);
    //free(cpu_rflowmtx);
    free(Excess_total);
    
    // return 0 and end program
    return 0;

}
